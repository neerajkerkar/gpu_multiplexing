#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <pthread.h>
#include "slicing.h"


// Kernel definition needs to have 2 extra fields blockOffset and realGridDim
// within kernel use blockOffset + blockId instead of blockId directly
// use realGridDim instead of gridDim
__global__ void MatAdd(dim3 blockOffset, dim3 realGridDim, int N, float* A, float* B, float* C)
{
    int rBlockIdx = blockOffset.x + blockIdx.x;
    int rBlockIdy = blockOffset.y + blockIdx.y;
    int rBlockIdz = blockOffset.z + blockIdx.z;

    int i = rBlockIdx * blockDim.x + threadIdx.x;
    int j = rBlockIdy * blockDim.y + threadIdx.y;
    int k = i+j*N;
    if (i < N && j < N)
        C[k] = A[k] + B[k];
}

// Kernel function to add the elements of two arrays
__global__
void VecAdd(dim3 blockOffset, dim3 realGridDim, int n, float *x, float *y)
{
  int rBlockIdx = blockOffset.x + blockIdx.x;
  int rBlockIdy = blockOffset.y + blockIdx.y;
  int rBlockIdz = blockOffset.z + blockIdx.z;

  int index = rBlockIdx * blockDim.x + threadIdx.x;
  int stride = blockDim.x * realGridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

__global__ void dummy_kernel(int *a){
  a+=1;
}

void* launch_matadd(void* dummy){
    int N = 1<<13;
    float *a,*b,*c;
  
    hipMallocManaged(&a, N*N*sizeof(float));
    hipMallocManaged(&b, N*N*sizeof(float));
    hipMallocManaged(&c, N*N*sizeof(float));

    for (int i = 0; i < N*N; i++) {
      a[i] = 1.0f;
      b[i] = 2.0f;
    }

    dim3 threadsPerBlock ={16, 16, 1};
    dim3 numBlocks={N / threadsPerBlock.x, N / threadsPerBlock.y, 1};
    
    //hipDeviceSynchronize();
    
    dim3 blockOff = {0,0,0};
    MatAdd<<<numBlocks, threadsPerBlock>>>(blockOff, numBlocks, N, a, b, c);

    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N*N; i++)
      maxError = fmax(maxError, fabs(c[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(a);
    hipFree(b);
    hipFree(c);
  
    return NULL;
}

void* launch_vecadd(void* dummy)
{
  int N = 1<<26;
  float *x, *y;
  

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  dim3 nBlocks = {numBlocks, 1, 1};

  dim3 blockOff = {0,0,0};
  VecAdd<<<nBlocks, blockSize>>>(blockOff, nBlocks, N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return NULL;
}

int main(void){
  const int num_threads = 2;
  pthread_t threads[num_threads];
  /*int a=1;
  dummy_kernel<<<1,1>>>(&a);
  dim3 blockOff = {0,0,0};
  dim3 grid = {0,0,0};
  MatAdd<<<1,1>>>(blockOff,grid,0,NULL,NULL,NULL);*/
  for (int i = 0; i < num_threads; i++) {
    void * (*launch_kernel)(void *);
    if(i%2==1) launch_kernel = launch_matadd;
    else launch_kernel = launch_vecadd;
    if (pthread_create(&threads[i], NULL, launch_kernel, 0)) {
      fprintf(stderr, "Error creating threadn");
      return 1;
    }
  }
  for (int i = 0; i < num_threads; i++) {
    if(pthread_join(threads[i], NULL)) {
      fprintf(stderr, "Error joining threadn");
      return 2;
    }
  }
  return 0;
}


